#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <time.h>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
using namespace std;



//Faltan
//Bomba
//tnt
//gravedad
//rompecabezas
//Automatico


//---------Método que comprueba si lo introducido por consola es un valor entero.---------
bool validarEntero(int& numero) {
	//Si no es un numero entero lo introducido, el código gestiona el error.
	if (cin.fail()) {
		cin.clear();
		cin.ignore();
		cout << "[Error]: El valor introducido no es correcto." << endl;
		cout << endl;
		return false;
	}
	else {
		cout << endl;
		return true;
	}
}

//---------Método que pide al usuario las filas para comenzar.---------
int pedirFilas() {
	int filas;
	bool validarEnteroCheck = false;

	while (!validarEnteroCheck) {
		cin >> filas;
		validarEnteroCheck = validarEntero(filas);
	}
	validarEnteroCheck = false;
	return filas;
}

//---------Método que pide al usuario las columnas para comenzar.---------
int pedirColumnas() {
	int columnas;
	bool validarEnteroCheck = false;
	//--------Pide las columnas de la matriz. Si no introduce un numero entero, error.------------
	while (!validarEnteroCheck) {
		cin >> columnas;
		validarEnteroCheck = validarEntero(columnas);
	}
	validarEnteroCheck = false;
	return columnas;
}

//---------Método que pide al usuario la dificultad del juego.---------
int pedirDificultad() {
	//Pide el nivel de dificultad. Si no introduce un numero entero o (1 o 2),Error.
	// FACIL = 1. DIFICIL = 2
	int dificultad;
	bool validarEnteroCheck = false;
	while (!validarEnteroCheck) {
		cout << "Selecciona nivel de dificultad (1 = FACIL | 2 = DIFICIL)" << endl;
		cout << "\tFACIL -> Candry Crosh genera numeros del 1 al 4." << endl;
		cout << "\tDIFICIL -> Candry Crosh genera numeros del 1 al 6." << endl;
		cout << "Introduce el numero: ";
		cin >> dificultad;
		validarEnteroCheck = validarEntero(dificultad);
		if (dificultad != 1 && dificultad != 2) { validarEnteroCheck = false; }
	}
	validarEnteroCheck = false;


	return dificultad;

}

//---------Método que genera la matriz inicial y la devuelve.---------
int* generarMatriz(int* h_matriz, int filas, int columnas, int dificultad, hiprandState* estado) {
	//Se genera la matriz
	printf("\n Da comienzo el juego\n");
	printf("**********************");
	printf("\n Se genera aleatoriamente la siguiente matriz:\n");
	for (int i = 0; i < filas; i++) {
		for (int j = 0; j < columnas; j++) {
			if (dificultad == 1) {
				h_matriz[i * columnas + j] = (rand() % 4) + 1;; //numero aleatorio entre el 1-4
				printf("%d", h_matriz[i * columnas + j]);
				printf("  ");
			}
			else if (dificultad == 2) {
				h_matriz[i * columnas + j] = (rand() % 6) + 1; //numero aleatorio entre el 1-6
				printf("%d", h_matriz[i * columnas + j]);
				printf("  ");
			}

		}
		printf("\n");
	}
	printf("**********************\n");

	return h_matriz;

}

//---------Método que genera la matriz inicial y la devuelve.---------
void imprimirMatriz2(int* h_R, int c, int f) {
	//Se genera la matriz

	printf("\n Tablero:\n");
	printf("---------\n");
	for (int i = 0; i < f; i++) {
		for (int j = 0; j < c; j++) {

			//printf("%d", dev_R[i*c+j]);
			//printf("  ");
			//switch (h_R[i * c + j]) {
			switch (h_R[i * c + j]) {
			case -3:
				printf("R");
				break;
			case -2:
				printf("T");
				break;
			case -1:
				printf("B");
				break;
			case 0:
				printf("O");
				break;
			case 1:
				printf("1");
				break;
			case 2:
				printf("2");
				break;
			case 3:
				printf("3");
				break;
			case 4:
				printf("4");
				break;
			case 5:
				printf("5");
				break;
			case 6:
				printf("6");
				break;
			}
			printf(" ");
		}
		printf("\n");
	}
	printf("---------\n");

}


//-----------------------------------------------------------

__device__ int* gravedad_horizontal_TNT(int* dev_R, int f, int c, int fila, int* arr_adyacente, int n, int dificultad, hiprandState* estado) {
	//fila actual de la posicion
	//int fila_actual = pos/c;
	int pos = arr_adyacente[0];
	int pos2 = arr_adyacente[1];
	int pos3 = arr_adyacente[2];
	int pos4 = arr_adyacente[3];
	int pos5 = arr_adyacente[4];
	int pos6 = arr_adyacente[5];
	if (fila == 0) {
		if (dificultad == 1) {
			dev_R[pos2] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos3] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos4] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos5] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos6] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos] = -2;
		}
		else {
			dev_R[pos2] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos3] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos4] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos5] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos6] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos] = -2;
		}
	}
	else {
		for (int i = 0; i < fila; i++) {
			//dev_R[pos-(c*i)] = dev_R[pos-(c*(i+1))];
			dev_R[pos2 - (c * i)] = dev_R[pos2 - (c * (i + 1))];
			dev_R[pos3 - (c * i)] = dev_R[pos3 - (c * (i + 1))];
			dev_R[pos4 - (c * i)] = dev_R[pos4 - (c * (i + 1))];
			dev_R[pos5 - (c * i)] = dev_R[pos5 - (c * (i + 1))];
			dev_R[pos6 - (c * i)] = dev_R[pos6 - (c * (i + 1))];
		}
		//Introduce los valores aleatoros de la primera fila

		if (dificultad == 1) {
			dev_R[pos2 % c] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos3 % c] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos4 % c] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos5 % c] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos6 % c] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos] = -2;
		}
		else {
			dev_R[pos2 % c] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos3 % c] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos4 % c] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos5 % c] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos6 % c] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos] = -2;
		}
		
	}
	return dev_R;
}

__device__ int* gravedad_vertical_TNT(int* dev_R, int f, int c, int fila, int* arr_adyacente, int n, int dificultad, hiprandState* estado) {

	int pos = arr_adyacente[0];
	int pos2 = arr_adyacente[1];
	int pos3 = arr_adyacente[2];
	int pos4 = arr_adyacente[3];
	int pos5 = arr_adyacente[4];
	int pos6 = arr_adyacente[5];
	//se obtiene cuando comienza los 4 numeros adyancentes
	int fila_ini = pos / c;
	//se obtiene en que fila termina los cuatro repetidos
	int fila_ult = pos6 / c;
	//printf("\nnn: %d %d", fila_ini, fila_ult);
	//se obtiene cuantos huecos se tienen que intercambiar 
	int huecos_cambio = f - (fila_ult - fila_ini + 1);
	//printf("\nHuecos: %d", huecos_cambio);
	//se obtiene cual es la columna en la que iteramos
	int primera_columna = pos % c;

	//Obtiene cuantos huecos se tienen que asignar de forma aleatoria
	int huecos_aleatorios = fila_ult - huecos_cambio;

	//Se comprueba si hay alguna posicion que esta en la primera fila.
	//Si la hay no gravedad y se asignan los valores aleatorio.
	if (pos / c == 0 || pos2 / c == 0 || pos3 / c == 0 || pos4 / c == 0 || pos5 / c == 0 || pos6 / c == 0) {
		if (dificultad == 1) {
			dev_R[pos2] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos3] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos4] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos5] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos6] = 1 + (int)(hiprand_uniform(estado) * 4);
			dev_R[pos6] = -2;
		}
		else {
			dev_R[pos2] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos3] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos4] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos5] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos6] = 1 + (int)(hiprand_uniform(estado) * 6);
			dev_R[pos6] = -2;
		}
	}
	else {
		//Cambia los numeros repetidos por los de arriba
		for (int i = 0; i < huecos_cambio; i++) {
			dev_R[pos5 - (c * i)] = dev_R[pos - (c * (i + 1))];
		}
		//Cambia los valores de arriba por valores aleatorios
		for (int i = 0; i < fila_ini; i++) {
			if (dificultad == 1) {
				dev_R[primera_columna + (c * i)] = 1 + (int)(hiprand_uniform(estado) * 4);
			}
			else {
				dev_R[primera_columna + (c * i)] = 1 + (int)(hiprand_uniform(estado) * 6);
			}
		}
		dev_R[pos6] = -2;
	}
	return dev_R;
}




__global__ void ponerTNT(int* dev_R, int c, int f, int size, int fila, int columna, int* TNT, int n, int dificultad, hiprandState* estado) {

	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;

	//posicion de la matriz final 
	int pos = fil * c + col;


	//posicion indicada por el usuario
	int posUsuario = fila * c + columna;
	int filaUsuario = posUsuario / c;

	//Array que contendra las posiciones adyacentes y el valor de cada posicion 
	__shared__ int valor[1];
	__shared__ int arr_adyacente[5];


	///
	//comprueba que se esta en la pos en la que esta interactuando el usuario
	if (pos == posUsuario) {

		valor[0] = dev_R[pos];
		valor[1] = pos;
		arr_adyacente[0] = pos;
		//printf("Posicionwes: %d %d %d %d %d %d", dev_R[pos - (c * 5)], dev_R[pos - (c * 4)], dev_R[pos - (c * 3)], dev_R[pos - (c * 2)], dev_R[pos - (c * 1)], dev_R[pos + c]);

	}
	if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 1] && valor[0] == dev_R[pos + 1] && valor[0] == dev_R[pos + 2] && valor[0] == dev_R[pos + 3] && valor[0] == dev_R[pos + 4] && pos / c == fila && (pos + 1) / c == fila && (pos + 2) / c == fila && (pos + 3) / c == fila && (pos + 4) / c == fila) {
		printf("Posicion que va a poner el TNT: %d %d", fil, col);
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos + 1; arr_adyacente[3] = pos + 2; arr_adyacente[4] = pos + 3; arr_adyacente[5] = pos + 4;
		dev_R = gravedad_horizontal_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;

	}
	else if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && valor[0] == dev_R[pos + 1] && valor[0] == dev_R[pos + 2] && valor[0] == dev_R[pos + 3] && pos / c == fila && (pos - 1) / c == fila && (pos + 1) / c == fila && (pos + 2) / c == fila && (pos + 3) / c == fila) {
		printf("Posicion que va a poner el TNT: %d %d", fil, col);
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos - 2; arr_adyacente[3] = pos + 1; arr_adyacente[4] = pos + 2; arr_adyacente[5] = pos + 3;
		dev_R = gravedad_horizontal_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;

	}
	else if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 3] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && valor[0] == dev_R[pos + 1] && valor[0] == dev_R[pos + 2] && pos / c == fila && (pos - 2) / c == fila && (pos - 1) / c == fila && (pos + 1) / c == fila && (pos - 3) / c == fila) {
		printf("Posicion que va a poner el TNT: %d %d", fil, col);
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos - 3; arr_adyacente[3] = pos - 2; arr_adyacente[4] = pos + 1; arr_adyacente[5] = pos + 2;
		dev_R = gravedad_horizontal_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;

	}
	else if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 4] && valor[0] == dev_R[pos - 3] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && valor[0] == dev_R[pos + 1] && pos / c == fila && (pos - 3) / c == fila && (pos - 2) / c == fila && (pos - 1) / c == fila && (pos - 4) / c == fila) {
		printf("Posicion que va a poner el TNT: %d %d", fil, col);
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos - 4; arr_adyacente[3] = pos - 3; arr_adyacente[4] = pos - 2; arr_adyacente[5] = pos + 1;
		dev_R = gravedad_horizontal_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;

	}
	else if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 5] && valor[0] == dev_R[pos - 4] && valor[0] == dev_R[pos - 3] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && pos / c == fila && (pos - 4) / c == fila && (pos - 3) / c == fila && (pos - 2) / c == fila && (pos - 5) / c == fila) {
		printf("Posicion que va a poner el TNT: %d %d", fil, col);
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos - 5; arr_adyacente[3] = pos - 4; arr_adyacente[4] = pos - 3; arr_adyacente[5] = pos - 2;
		dev_R = gravedad_horizontal_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;

	}
	else if (valor[1] == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 5] && valor[0] == dev_R[pos - 4] && valor[0] == dev_R[pos - 3] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && pos / c == fila && (pos - 4) / c == fila && (pos - 3) / c == fila && (pos - 2) / c == fila && (pos - 5) / c == fila) {

		printf("\nHola3\n");
		printf("Posicion que va a poner el TNT: %d ", pos);
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos - 5; arr_adyacente[3] = pos - 4; arr_adyacente[4] = pos - 3; arr_adyacente[5] = pos - 2;
		dev_R = gravedad_horizontal_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;

	}
	//Vertical
	if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - c] && valor[0] == dev_R[pos + c] && valor[0] == dev_R[pos + (c * 2)] && valor[0] == dev_R[pos + (c * 3)] && valor[0] == dev_R[pos + (c * 4)]) {
		printf("Posicion que va a poner el TNT6: %d %d", fil, col);
		arr_adyacente[0] = pos - c; arr_adyacente[1] = pos; arr_adyacente[2] = pos + c; arr_adyacente[3] = pos + (c * 2); arr_adyacente[4] = pos + (c * 3); arr_adyacente[5] = pos + (c * 4);
		dev_R = gravedad_vertical_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;
	}
	if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - c] && valor[0] == dev_R[pos + c] && valor[0] == dev_R[pos + (c * 2)] && valor[0] == dev_R[pos + (c * 3)]) {
		printf("Posicion que va a poner el TNT7: %d %d", fil, col);
		arr_adyacente[0] = pos - (c * 2); arr_adyacente[1] = pos - c; arr_adyacente[2] = pos; arr_adyacente[3] = pos + c; arr_adyacente[4] = pos + (c * 2); arr_adyacente[5] = pos + (c * 3);
		dev_R = gravedad_vertical_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;
	}
	if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 3)] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - c] && valor[0] == dev_R[pos + c] && valor[0] == dev_R[pos + (c * 2)]) {
		printf("Posicion que va a poner el TNT8: %d %d", fil, col);
		arr_adyacente[0] = pos - (c * 3); arr_adyacente[1] = pos - (c * 2); arr_adyacente[2] = pos - c; arr_adyacente[3] = pos; arr_adyacente[4] = pos + c; arr_adyacente[5] = pos + (c * 2);
		dev_R = gravedad_vertical_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;
	}
	if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 4)] && valor[0] == dev_R[pos - (c * 3)] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - c] && valor[0] == dev_R[pos + c]) {
		printf("Posicion que va a poner el TNT9: %d %d", fil, col);
		arr_adyacente[0] = pos - (c * 4); arr_adyacente[1] = pos - (c * 3); arr_adyacente[2] = pos - (c * 2); arr_adyacente[3] = pos - c; arr_adyacente[4] = pos; arr_adyacente[5] = pos + c;
		dev_R = gravedad_vertical_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;
	}
	if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 5)] && valor[0] == dev_R[pos - (c * 4)] && valor[0] == dev_R[pos - (c * 3)] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - c]) {
		printf("Posicion que va a poner el TNT10: %d %d", fil, col);
		arr_adyacente[0] = pos - (c * 5); arr_adyacente[1] = pos - (c * 4); arr_adyacente[2] = pos - (c * 3); arr_adyacente[3] = pos - (c * 2); arr_adyacente[4] = pos - c; arr_adyacente[5] = pos;
		dev_R = gravedad_vertical_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;
	} if (valor[1] == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 5)] && valor[0] == dev_R[pos - (c * 4)] && valor[0] == dev_R[pos - (c * 3)] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - c]) {

		printf("Posicion que va a poner el TNT11: %d %d", fil, col);
		arr_adyacente[0] = pos - (c * 5); arr_adyacente[1] = pos - (c * 4); arr_adyacente[2] = pos - (c * 3); arr_adyacente[3] = pos - (c * 2); arr_adyacente[4] = pos - c; arr_adyacente[5] = pos;
		dev_R = gravedad_vertical_TNT(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*TNT += 1;
	}
	__syncthreads();
}

__global__ void explotarTNT(int* dev_R, int c, int f, int size, int fila, int columna, int n, int* explosion, int dificultad, hiprandState* estado) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;

	//posicion de la matriz 
	int pos = fil * c + col;
	//posicion indicada por el usuario
	int posUsuario = fila * c + columna;
	int col_final = pos;
	int fil_inicial = pos;
	//pos inicial va empezar el pos 
	int pos_ini = pos;


	if (pos == posUsuario && dev_R[pos] == -2) {
		printf("Se explota el TNT, se explotan todas las casillas en un radio de 4");

		for (int i = 0; i < 4; i++) {
			//printf("\nPOS: %d %d", pos_ini, fila);
			//se comprueba que la posicion mas a la izq esta en la misma fila

			if ((pos_ini - 1) / c == fila && pos_ini - 1 >= 0) {
				pos_ini = pos_ini - 1;
			}
			//se comprueba que la posicion mas a la dere esta en la misma fila
			if (col_final / c == fila) {
				col_final = col_final + 1;
			}
		}
		//printf("\nPosicion inicial1: %d", pos_ini);
		for (int i = 0; i < 4; i++) {
			//printf("\nPOS: %d %d", pos_ini, fila);
			//se comprueba que la posicion mas abajo esta dentro del tablero
			if (pos_ini / c == f - 1) {
				pos_ini = pos_ini;
			}
			else {
				pos_ini = pos_ini + c;
			}
			//se calcula cual es la fila inicial
			if (fil_inicial / c == 0) {
				fil_inicial = fil_inicial;
			}
			else {
				fil_inicial = fil_inicial - c;
			}
		}
		//printf("\nPosicion inicial: %d", pos_ini);

		//se tiene que comprobar cual es la diferencia entre la menor fila a la que puede llegar al explosion y a la mas arriba que puede, ese sera el numero del for
		int fila_ult = pos_ini / c;
		int diferencia_fila = fila_ult - (fil_inicial / c);
		//se comprueba en que columna empieza y cual acaba
		int col_primera = pos_ini % c;
		//printf("\nColumnas: %d %d",col_final%c, col_primera);
		int diferencia_columna = abs((col_final % c) - col_primera);

		//printf("\ndiferencia_fila: %d", diferencia_fila);
		//printf("\ndiferencia_columna: %d", diferencia_columna);

		int col_restar = c * diferencia_fila;

		//Se intercambian los valores de arriba de la explosion del TNT en las posiciones que se han explotado
		for (int i = 0; i < diferencia_columna + 1; i++) {
			for (int j = 0; j < diferencia_fila + 1; j++) {
				//printf("\nFilaa: %d", pos_ini - (c * j) + i);
				//printf("Cambio: %d ", pos_ini - ((c * (j + 1)) + col_restar) + i);
				dev_R[pos_ini - (c * j) + i] = dev_R[pos_ini - ((c * (j + 1)) + col_restar) + i];
			}

		}
		//se calcula cuantos elemenos de la fila se ponen de forma alearoia
		//int huecos_aleatorios = c-col_primera;
		int huecos_aleatorios = 0;
		int nueva_col_primera = col_primera;
		for (int i = 0; i < 9; i++) {
			//printf("\ncool: %d", nueva_col_primera);
			//printf("\nhuecos: %d", huecos_aleatorios);
			if (huecos_aleatorios < c - 1 && huecos_aleatorios <= 4 + columna && nueva_col_primera + 1 <= c) {
				huecos_aleatorios = huecos_aleatorios + 1;
				nueva_col_primera = nueva_col_primera + 1;

			}
		}

		//se cambiam las filas de arriba por valores aleatorios
		//printf("\nhuecos: %d", huecos_aleatorios);
		//printf("\nfilas: %d", diferencia_fila + 1);
		for (int i = 0; i < huecos_aleatorios; i++) {
			for (int j = 0; j < diferencia_fila + 1; j++) {
				//printf("\nPossiciones nuevas: %d", col_primera + (c * j) + i);

				if (dificultad == 1) {
					dev_R[col_primera + (c * j) + i] = 1 + (int)(hiprand_uniform(estado) * 4);
				}
				else {
					dev_R[col_primera + (c * j) + i] = 1 + (int)(hiprand_uniform(estado) * 6);
				}
			}
		}
		*explosion += 1;
	}

}




__device__ int* asignarAleatorios(int* dev_R, int pos, int pos2, int pos3, int pos4, int pos5, int n, int dificultad, hiprandState* estado) {

	/*
	dev_R[pos + 1] = 1 + (int)(hiprand_uniform(estado) * n);
	dev_R[pos + 2] = 1 + (int)(hiprand_uniform(estado) * n);
	dev_R[pos + 3] = 1 + (int)(hiprand_uniform(estado) * n);
	*/
	if (dificultad == 1) {
		dev_R[pos2] = 1 + (int)(hiprand_uniform(estado) * 4);
		dev_R[pos3] = 1 + (int)(hiprand_uniform(estado) * 4);
		dev_R[pos4] = 1 + (int)(hiprand_uniform(estado) * 4);
		dev_R[pos5] = 1 + (int)(hiprand_uniform(estado) * 4);
	}
	else {
		dev_R[pos2] = 1 + (int)(hiprand_uniform(estado) * 6);
		dev_R[pos3] = 1 + (int)(hiprand_uniform(estado) * 6);
		dev_R[pos4] = 1 + (int)(hiprand_uniform(estado) * 6);
		dev_R[pos5] = 1 + (int)(hiprand_uniform(estado) * 6);
	}
	return dev_R;
}

__device__ int* gravedad_vertical_bomba(int* dev_R, int f, int c, int fila, int* arr_adyacente, int n, int dificultad, hiprandState* estado) {
	
	int pos = arr_adyacente[0];
	int pos2 = arr_adyacente[1];
	int pos3 = arr_adyacente[2];
	int pos4 = arr_adyacente[3];
	int pos5 = arr_adyacente[4];
	hiprand_init(1234, pos, 0, estado);
	//se obtiene cuando comienza los 4 numeros adyancentes
	int fila_ini = pos / c;
	//se obtiene en que fila termina los cuatro repetidos
	int fila_ult = pos5 / c;
	//se obtiene cuantos huecos se tienen que poner de forma aleatoria
	int filas_aleatorio = f - (fila_ult - fila_ini);
	//printf("\nHuecos: %d", filas_aleatorio);
	//se obtiene cual es la columna en la que iteramos
	int primera_columna = pos % c;

	//Se comprueba si hay alguna posicion que esta en la primera fila.
	//Si la hay no gravedad y se asignan los valores aleatorio.
	if (pos / c == 0 || pos2 / c == 0 || pos3 / c == 0 || pos4 / c == 0 || pos5 / c == 0) {

		dev_R = asignarAleatorios(dev_R, pos, pos2, pos3, pos4, pos5, n, dificultad, estado);
		dev_R[pos] = -1;
	}
	else {
		//Cambia los numeros repetidos por los de arriba
		for (int i = 0; i < fila; i++) {
			//printf("\niteraciones1: %d", i);
			dev_R[pos4 - (c * i)] = dev_R[pos - (c * (i + 1))];
		}
		//Cambia los valores de arriba por valores aleatorios
		for (int i = 0; i < fila_ini-1; i++) {
			if (dificultad == 1) {
				dev_R[primera_columna + (c * i)] = 1 + (int)(hiprand_uniform(estado) * 4);
			}
			else {
				dev_R[primera_columna + (c * i)] = 1 + (int)(hiprand_uniform(estado) * 6);
			}

			
		}
		dev_R[pos5] = -1;
	}

	return dev_R;


}

__device__ int* gravedad_horizontal_bomba(int* dev_R, int f, int c, int fila, int* arr_adyacente, int n, int dificultad, hiprandState* estado) {
	//fila actual de la posicion
	//int fila_actual = pos/c;

	int pos = arr_adyacente[0];
	int pos2 = arr_adyacente[1];
	int pos3 = arr_adyacente[2];
	int pos4 = arr_adyacente[3];
	int pos5 = arr_adyacente[4];

	if (fila == 0) {
		dev_R = asignarAleatorios(dev_R, pos, pos2, pos3, pos4, pos5, n, dificultad, estado);
		dev_R[pos] = -1;
	}
	else {

		for (int i = 0; i < fila; i++) {
			//dev_R[pos-(c*i)] = dev_R[pos-(c*(i+1))];
			dev_R[pos2 - (c * i)] = dev_R[pos2 - (c * (i + 1))];
			dev_R[pos3 - (c * i)] = dev_R[pos3 - (c * (i + 1))];
			dev_R[pos4 - (c * i)] = dev_R[pos4 - (c * (i + 1))];
			dev_R[pos5 - (c * i)] = dev_R[pos5 - (c * (i + 1))];
		}
		//Introduce los valores aleatoros de la primera fila
		dev_R = asignarAleatorios(dev_R, pos % c, pos2 % c, pos3 % c, pos4 % c, pos5 % c, n, dificultad, estado);
		dev_R[pos] = -1;
	}

	return dev_R;
}


//se busca donde haya cuatro elementos iguales, tanto en posiciones horizontales como verticales y se pone la bomba
//se tiene que guardar si es columna o fila, para despues explotar

//---------Método que pone la bomba donde haya 5 elementos iguales, indicados por el usuario.---------
__global__ void ponerBomba(int* dev_R, int c, int f, int size, int fila, int columna, int* bomba, int n, int dificultad, hiprandState* estado) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;

	//posicion de la matriz final 
	int pos = fil * c + col;

	//posicion indicada por el usuario
	int posUsuario = fila * c + columna;
	int filaUsuario = posUsuario / c;

	//Array que contendra las posiciones adyacentes y el valor de cada posicion 
	__shared__ int valor[1];
	__shared__ int arr_adyacente[4];


	///
	//comprueba que se esta en la pos en la que esta interactuando el usuario
	if (pos == posUsuario) {

		valor[0] = dev_R[pos];
		valor[1] = pos;
		arr_adyacente[0] = pos;

	}

	__syncthreads();

	if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 1] && valor[0] == dev_R[pos + 1] && valor[0] == dev_R[pos + 2] && valor[0] == dev_R[pos + 3] && pos / c == fila && (pos + 1) / c == fila && (pos + 2) / c == fila && (pos + 3) / c == fila) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos + 1; arr_adyacente[3] = pos + 2; arr_adyacente[4] = pos + 3;
		dev_R = gravedad_horizontal_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}

	if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && valor[0] == dev_R[pos + 1] && valor[0] == dev_R[pos + 2] && (pos - 1) / c == fila && (pos + 1) / c == fila && (pos + 2) / c == fila) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos - 2; arr_adyacente[3] = pos + 1; arr_adyacente[4] = pos + 2;
		dev_R = gravedad_horizontal_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}
	if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 3] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && valor[0] == dev_R[pos + 1] && (pos - 2) / c == fila && (pos - 1) / c == fila && (pos + 1) / c == fila) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos - 3; arr_adyacente[3] = pos - 2; arr_adyacente[4] = pos + 1;
		dev_R = gravedad_horizontal_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}
	if (pos != posUsuario && valor[1] + 1 == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 4] && valor[0] == dev_R[pos - 3] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && (pos - 3) / c == fila && (pos - 2) / c == fila && (pos - 1) / c == fila) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - 1; arr_adyacente[1] = pos; arr_adyacente[2] = pos - 4; arr_adyacente[3] = pos - 3; arr_adyacente[4] = pos - 2;
		dev_R = gravedad_horizontal_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}
	if (valor[1] == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - 4] && valor[0] == dev_R[pos - 3] && valor[0] == dev_R[pos - 2] && valor[0] == dev_R[pos - 1] && (pos - 3) / c == fila && (pos - 2) / c == fila && (pos - 1) / c == fila) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos; arr_adyacente[1] = pos - 1; arr_adyacente[2] = pos - 2; arr_adyacente[3] = pos - 3; arr_adyacente[4] = pos - 4;
		dev_R = gravedad_horizontal_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}

	//En vertical
	//compruba que los 3 elementos de arriba tienen el mismo valor
	if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - c] && valor[0] == dev_R[pos + c] && valor[0] == dev_R[pos + (c * 2)] && valor[0] == dev_R[pos + (c * 3)]) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - c; arr_adyacente[1] = pos; arr_adyacente[2] = pos + c; arr_adyacente[3] = pos + (c * 2); arr_adyacente[4] = pos + (c * 3);
		dev_R = gravedad_vertical_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}
	else if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - c] && valor[0] == dev_R[pos + c] && valor[0] == dev_R[pos + (c * 2)]) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - (c * 2); arr_adyacente[1] = pos - c; arr_adyacente[2] = pos; arr_adyacente[3] = pos + c; arr_adyacente[4] = pos + (c * 2);
		dev_R = gravedad_vertical_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}
	else if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 3)] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - c] && valor[0] == dev_R[pos + c]) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - (c * 3); arr_adyacente[1] = pos - (c * 2); arr_adyacente[2] = pos - c; arr_adyacente[3] = pos; arr_adyacente[4] = pos + c;
		dev_R = gravedad_vertical_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}
	else if (pos != posUsuario && valor[1] + c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 4)] && valor[0] == dev_R[pos - (c * 3)] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - c]) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - (c * 4); arr_adyacente[1] = pos - (c * 3); arr_adyacente[2] = pos - (c * 2); arr_adyacente[3] = pos - c; arr_adyacente[4] = pos;
		dev_R = gravedad_vertical_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;

	}
	else if (pos != posUsuario && valor[1] - c == pos && valor[0] == dev_R[pos] && valor[0] == dev_R[pos - (c * 3)] && valor[0] == dev_R[pos - (c * 2)] && valor[0] == dev_R[pos - (c * 1)] && valor[0] == dev_R[pos + c]) {
		printf("Bomba Puesta \n");
		arr_adyacente[0] = pos - (c * 3); arr_adyacente[1] = pos - (c * 2); arr_adyacente[2] = pos - (c * 1); arr_adyacente[3] = pos; arr_adyacente[4] = pos + c;
		dev_R = gravedad_vertical_bomba(dev_R, f, c, fila, arr_adyacente, n, dificultad, estado);
		*bomba += 1;
	}
	__syncthreads();

}

__global__ void explotarBomba(int* dev_R, int c, int f, int size, int fila, int columna, int n, int* explosion, int dificultad, hiprandState* estado) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;

	//posicion de la matriz 
	int pos = fil * c + col;
	//posicion indicada por el usuario
	int posUsuario = fila * c + columna;
	//posicion de comienzo de la fila (fila por columnas totales de la matriz)
	int ini_fila = fila * c;
	//posicion de comienzo de la columna (columna por filas totales de la matriz)
	int ini_columna = columna * c;
	//numero aleatorio entre 0 y 1 que indica si se explota la fila o la columna
	//int f_c = rand() % 2;
	int f_c = 0;

	hiprand_init(1234, pos, 0, estado);


	if (pos == posUsuario && dev_R[pos] == -1) {
		//explota fila
		printf("\n [Informacion] Se explota la bomba\n");
		if (f_c == 0) {
			printf("Se genera una nueva fila y caen las demas: \n");
			//se cambian las cada una de las posiciones por la de arriba
			for (int i = 0; i < c; i++) {
				for (int j = 0; j < fila; j++) {
					//printf("\nFilaa: %d",ini_fila-(c*j)+ i);
					//printf("Cambio: %d ",ini_fila-(c*(j+1))+ i );
					dev_R[ini_fila - (c * j) + i] = dev_R[ini_fila - (c * (j + 1)) + i];
				}
			}
			//se cambia la primera fila por valores aleatorios
			for (int i = 0; i < c; i++) {
				
				if (dificultad == 1) {
					dev_R[0 + i] = 1 + (int)(hiprand_uniform(estado) * 4);
				}
				else {
					dev_R[0 + i] = 1 + (int)(hiprand_uniform(estado) * 6);
				}
			}

		}
		else {
			printf("Se genera una nueva columna: \n");
			//explota la columna
			//se cambia toda esa columna por valores aleatorios
			for (int i = 0; i < f; i++) {
				if (dificultad == 1) {
					dev_R[ini_columna + (c * i)] = 1 + (int)(hiprand_uniform(estado) * 4);
				}
				else {
					dev_R[ini_columna + (c * i)] = 1 + (int)(hiprand_uniform(estado) * 6);
				}
			}
		}
		*explosion += 1;
	}
}



//Chequea si hay elementos adyacentes con el mismo valor en toda la matriz, si es asi, los cambia por valores aleatorios
__global__ void comprobarPares(int* dev_R, int c, int f, int size, int fila, int columna, int* par, int dificultad, hiprandState* estado) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;

	//posicion de la matriz final 
	int pos = fil * c + col;
	//posicion indicada por el usuario
	int posUsuario = fila * c + columna;

	//Numeros aleatorios
	hiprand_init(1234, pos, 0, estado);

	//Array que contendrá las posiciones adyacentes
	__shared__ int valor[2];
	__shared__ int arr_adyacente[2];

	if (pos == posUsuario) {
		valor[0] = dev_R[pos];
		valor[1] = pos;
		arr_adyacente[0] = pos;
	}
	__syncthreads();
	//Comprobar arriba
	if ((valor[1]-c) == (pos) && valor[0] == dev_R[pos] && pos != posUsuario) {
		arr_adyacente[1] = pos;
		*par += 1;
	}
	//Comprobar abajo
	else if ((valor[1] + c) == (pos) && valor[0] == dev_R[pos] && pos != posUsuario) {
		arr_adyacente[1] = pos;
		*par += 1;
	}
	//Comprobar derecha
	else if ((valor[1] + 1) == (pos) && valor[0] == dev_R[pos] && pos != posUsuario) {
		arr_adyacente[1] = pos;
		*par += 1;
	}
	//Comprueba izquierda
	else if ((valor[1] - 1) == (pos) && valor[0] == dev_R[pos] && pos != posUsuario) {
		arr_adyacente[1] = pos;
		*par += 1;
	}
	__syncthreads();

	
	if (fil < f && col < c && pos == posUsuario && (dev_R[arr_adyacente[1]] == 1 || dev_R[arr_adyacente[1]] == 2 || dev_R[arr_adyacente[1]] == 3 || dev_R[arr_adyacente[1]] == 4 || dev_R[arr_adyacente[1]] == 5 || dev_R[arr_adyacente[1]] == 6)) {
		//Arriba
		if (pos == arr_adyacente[1] + c) {
			int nuevo_f0 = 0; //Valor Declarativo
			int nuevo_f1 = 0; //Valor Declarativo
			//printf("\nSe añaden los valores: ");
			if (dificultad == 1) { //Coge aleatorios los nuevos valores de la fila 0 y fila 1
				nuevo_f0 = 1 + (int)(hiprand_uniform(estado) * 4);
				nuevo_f1 = 1 + (int)(hiprand_uniform(estado) * 4);
			}
			else {
				nuevo_f0 = 1 + (int)(hiprand_uniform(estado) * 6);
				nuevo_f1 = 1 + (int)(hiprand_uniform(estado) * 6);
			}
			int kAux = 2;
			int kFinal = 0;
			for (int k = 0; k < fila - 1; k++) { //Rellena todos los valores de la columna por gravedad
				dev_R[pos - (c * k)] = dev_R[pos - (c * kAux)];
				kAux++;
				//printf("%d ", dev_R[pos - (c * k)]);
				kFinal = k;
			}
			kFinal++;
			dev_R[pos - (c * kFinal)] = nuevo_f1; //Rellena valor de la fila 1
			//printf("%d ", dev_R[pos - (c * kFinal)]);
			kFinal++;
			dev_R[pos - (c * kFinal)] = nuevo_f0; //Rellena valor de la fila 0
			//printf("%d\n", dev_R[pos - (c * kFinal)]);

			*par += 1;
		}
		//Abajo
		else if (pos == arr_adyacente[1] - c) {
			int nuevo_f0 = 0; //Valor Declarativo
			int nuevo_f1 = 0; //Valor Declarativo
			//printf("\nSe añaden los valores: ");
			if (dificultad == 1) { //Coge aleatorios los nuevos valores de la fila 0 y fila 1
				nuevo_f0 = 1 + (int)(hiprand_uniform(estado) * 4);
				nuevo_f1 = 1 + (int)(hiprand_uniform(estado) * 4);
			}
			else {
				nuevo_f0 = 1 + (int)(hiprand_uniform(estado) * 6);
				nuevo_f1 = 1 + (int)(hiprand_uniform(estado) * 6);
			}
			if (fila == 0) { //Si el item examinado esta en la fila 0, solo hay que generar aleatoriamente los valores de f0 y f1.
				dev_R[pos] = nuevo_f0;
				dev_R[pos + c] = nuevo_f1;
				//printf("%d ", dev_R[pos]);
				//printf("%d ", dev_R[pos + c]);
			}
			else { //Si el item examinado NO esta en la fila 0, hay que generar aleatoriamente los valores de f0 y f1 y el resto por gravedad.
				pos = pos + c; //Para que se comporte como el algortimo que aplica gravedad arriba
				int kAux = 2;
				int kFinal = 0;
				for (int k = 0; k < fila; k++) { //Rellena todos los valores de la columna por gravedad
					dev_R[pos - (c * k)] = dev_R[pos - (c * kAux)];
					kAux++;
					//printf("%d ", dev_R[pos - (c * k)]);
					kFinal = k;
				}
				kFinal++;
				dev_R[pos - (c * kFinal)] = nuevo_f1; //Rellena valor de la fila 1
				//printf("%d ", dev_R[pos - (c * kFinal)]);
				kFinal++;
				dev_R[pos - (c * kFinal)] = nuevo_f0; //Rellena valor de la fila 0
				//printf("%d\n", dev_R[pos - (c * kFinal)]);
			}
			*par += 1;
		}
		//Izquierda y derecha
		else if (pos == arr_adyacente[1] + 1 || pos == arr_adyacente[1]-1) {
			for (int i = 0; i < fila; i++) {
				dev_R[pos - (c * i)] = dev_R[pos - c * (i + 1)];
				dev_R[arr_adyacente[1] - (c * i)] = dev_R[arr_adyacente[1] - (c * (i + 1))];
				if (i == fila - 1) {
					if (dificultad == 1) {
						dev_R[pos - (c * (i+1))] = 1 + (int)(hiprand_uniform(estado) * 4);
						dev_R[arr_adyacente[1] - (c * (i + 1))] = 1 + (int)(hiprand_uniform(estado) * 4);
					}
					else {
						dev_R[pos - (c * (i + 1))] = 1 + (int)(hiprand_uniform(estado) * 6);
						dev_R[arr_adyacente[1] - (c * (i + 1))] = 1 + (int)(hiprand_uniform(estado) * 6);
					}
				}

			}
			par += 1;
		}
				
	}
	__syncthreads();
	
}

__global__ void comprobarRompecabezas(int* dev_R, int c, int f, int size, int fila, int columna, int* rompe, int dificultad, hiprandState* estado) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;

	//posicion de la matriz final 
	int pos = fil * c + col;
	//posicion indicada por el usuario
	int posUsuario = fila * c + columna;

	//Numeros aleatorios
	hiprand_init(1234, pos, 0, estado);

	int encontrado = 0;
	__shared__ int list_pos[6];
	__shared__ int list_pos2[6];
	if (pos == posUsuario) {
		//Rompecabezas en fila
		if (encontrado == 0) {
			int cumple_condicion = 1; //tiene que ser 7 para cumplir
			int pos_aux = pos + 1;
			int arr_aux = 0;
			//Cuenta cuantos bloques del mismo color tiene a la derecha
			while (dev_R[pos] == dev_R[pos_aux] && cumple_condicion < 7 && pos_aux < (((c + 1) * fila) + c + 1)) {
				list_pos[arr_aux] = pos_aux;
				pos_aux++;
				cumple_condicion++;
				arr_aux++;
			}
			__syncthreads();
			pos_aux = pos - 1;
			//Cuenta cuantos bloques del mismo color tiene a la izquierda
			while (dev_R[pos] == dev_R[pos_aux] && cumple_condicion < 7 && pos_aux >= (c * fila)) {
				list_pos[arr_aux] = pos_aux;
				pos_aux--;
				cumple_condicion++;
				arr_aux++;
			}
			__syncthreads();
			//Ha encontrado un rompecabezas y hay que ponerlos
			if (cumple_condicion == 7) {
				//gravedad si el rompecabezas esta en la fila 0
				if (fila == 0) {
					dev_R[pos] = -3;
					for (int k = 0; k < 6; k++) {
						if (dificultad == 1) {
							dev_R[list_pos[k]] = 1 + (int)(hiprand_uniform(estado) * 4);
						}
						else {
							dev_R[list_pos[k]] = 1 + (int)(hiprand_uniform(estado) * 6);
						}
					}
					__syncthreads();
				}
				//Gravedad si el rompecabezas esta en la fila != 0
				else {
					dev_R[pos] = -3;
					//Iteraciones que va a tener que hacer sobre las diferentes filas para cambiar los bloques
					for (int k = 0; k < fila; k++) {
						for (int m = 0; m < 6; m++) {
							dev_R[list_pos[m] - (c * k)] = dev_R[list_pos[m] - (c * (k + 1))];
						}
						__syncthreads();
						//Rellena primera fila de aleatorios
						if (k == fila - 1) {
							for (int m = 0; m < 6; m++) {
								if (dificultad == 1) {
									dev_R[list_pos[m] - (c * (k + 2))] = 1 + (int)(hiprand_uniform(estado) * 4);
								}
								else {
									dev_R[list_pos[m] - (c * (k + 2))] = 1 + (int)(hiprand_uniform(estado) * 6);
								}
							}
						}
						__syncthreads();
					}
				}
				encontrado = 1;
				*rompe += 1;
			}
		}
		//Rompecabezas en columna
		if (encontrado == 0) {
			int lista_ordenada[7];
			int cumple_condicion = 1; //tiene que ser 7 para cumplir
			int pos_aux = pos - c;
			int arr_aux = 0;
			//Cuenta bloques arriba
			while (dev_R[pos] == dev_R[pos_aux] && cumple_condicion < 7 && pos_aux > 0) {
				list_pos2[arr_aux] = pos_aux;
				pos_aux -= c;
				cumple_condicion++;
				arr_aux++;
			}
			__syncthreads();
			//Cuenta bloques abajo
			pos_aux = pos + c;
			while (dev_R[pos] == dev_R[pos_aux] && cumple_condicion < 7 && pos_aux < (f * c)) {
				list_pos2[arr_aux] = pos_aux;
				pos_aux += c;
				cumple_condicion++;
				arr_aux++;
			}
			__syncthreads();
			//Cumple condidicion
			if (cumple_condicion == 7) {
				for (int k = 0; k < 6; k++) {
					lista_ordenada[k] = list_pos2[k];
				}
				__syncthreads();
				lista_ordenada[6] = pos;

				for (int i = 0; i < 6; i++) {
					for (int j = 0; j < 6 - i; j++) {
						if (lista_ordenada[j] > lista_ordenada[j + 1]) {
							int temp = lista_ordenada[j];
							lista_ordenada[j] = lista_ordenada[j + 1];
							lista_ordenada[j + 1] = temp;
						}
					}
				}
				dev_R[lista_ordenada[6]] = -3;
				//Rellenar todo de numeros aleatorios
				if (lista_ordenada[0] <= c) {
					if (dificultad == 1) {
						dev_R[lista_ordenada[0]] = 1 + (int)(hiprand_uniform(estado) * 4);
						dev_R[lista_ordenada[1]] = 1 + (int)(hiprand_uniform(estado) * 4);
						dev_R[lista_ordenada[2]] = 1 + (int)(hiprand_uniform(estado) * 4);
						dev_R[lista_ordenada[3]] = 1 + (int)(hiprand_uniform(estado) * 4);
						dev_R[lista_ordenada[4]] = 1 + (int)(hiprand_uniform(estado) * 4);
						dev_R[lista_ordenada[5]] = 1 + (int)(hiprand_uniform(estado) * 4);
					}
					else {
						dev_R[lista_ordenada[0]] = 1 + (int)(hiprand_uniform(estado) * 6);
						dev_R[lista_ordenada[1]] = 1 + (int)(hiprand_uniform(estado) * 6);
						dev_R[lista_ordenada[2]] = 1 + (int)(hiprand_uniform(estado) * 6);
						dev_R[lista_ordenada[3]] = 1 + (int)(hiprand_uniform(estado) * 6);
						dev_R[lista_ordenada[4]] = 1 + (int)(hiprand_uniform(estado) * 6);
						dev_R[lista_ordenada[5]] = 1 + (int)(hiprand_uniform(estado) * 6);
					}
				}
				//Aplicar gravedad
				else {
					int posicionAux;
					posicionAux = lista_ordenada[0] - c;
					int iteraciones = 0;
					int iterador = 5;
					while (posicionAux > 0) {
						dev_R[lista_ordenada[iterador]] = dev_R[posicionAux];
						iteraciones += 1;
						posicionAux -= c;
						iterador -= 1;
					}
					for (int k = 0; k <= iterador; k++) {
						if (dificultad == 1) {
							dev_R[lista_ordenada[k]] = 1 + (int)(hiprand_uniform(estado) * 4);
						}
						else {
							dev_R[lista_ordenada[k]] = 1 + (int)(hiprand_uniform(estado) * 6);
						}
					}
					int posicionAux2;
					posicionAux2 = lista_ordenada[0] - c;
					while (posicionAux2 > 0) {
						if (dificultad == 1) {
							dev_R[posicionAux2] = 1 + (int)(hiprand_uniform(estado) * 4);
						}
						else {
							dev_R[posicionAux2] = 1 + (int)(hiprand_uniform(estado) * 6);
						}
						posicionAux2 -= c;
					}

				}
				encontrado = 1;
				*rompe += 1;
			}
		}
	}
	
}

__global__ void explotarRompecabezas(int* dev_R, int c, int f, int size, int fila, int columna, int* rompe, int dificultad, hiprandState* estado) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;

	//posicion de la matriz final 
	int pos = fil * c + col;
	//posicion indicada por el usuario
	int posUsuario = fila * c + columna;

	//Numeros aleatorios
	hiprand_init(1234, pos, 0, estado);

	int numero_quitar;
	if (dificultad == 1) {
		numero_quitar = 1 + (int)(hiprand_uniform(estado) * 4);
	}
	else {
		numero_quitar = 1 + (int)(hiprand_uniform(estado) * 6);
	}
	
	if (pos == posUsuario && dev_R[pos] == -3) {
		printf("\nExplotando el Rompecabezas con el color: %d \n", numero_quitar);
		//LO PRIMERO ES QUITAR EL BLOQUE ROMPECABEZAS Y APLICAR GRAVEDAD
		int ff = 0;
		for (int s = pos; s > c; s= s - c) {
			dev_R[s] = dev_R[s - c];
			ff = s;
		}
		if (dificultad == 1) {
			dev_R[ff-c] = 1 + (int)(hiprand_uniform(estado) * 4);
		}
		else {
			dev_R[ff-c] = 1 + (int)(hiprand_uniform(estado) * 6);
		}
		*rompe += 1;
		//for (int k=1; k <= c; k++) { CAMBIAR
		for (int k=1; k <= c; k++) {
			int valor_inicio = f * c - k;
			if (valor_inicio - c < 0) {//Solo hay una fila, asi que se pone aleatorio
				if (dificultad == 1) {
					dev_R[valor_inicio] = 1 + (int)(hiprand_uniform(estado) * 4);
				}
				else {
					dev_R[valor_inicio] = 1 + (int)(hiprand_uniform(estado) * 6);
				}
			}
			else { //Hay mas de de una fila, hay que evaluar valor a valor la columna
				int valor_evaluado = valor_inicio;
				int flag = 0;
				while (valor_evaluado > 0 && flag == 0) { //Se evaluan todos los valores, empezando desel el primero de la columna
					if (valor_evaluado < c) { //Se esta evaluando el ultimo valor
						if (dev_R[valor_evaluado] == numero_quitar) {//Si el ultimo  valor, coincide con el que se quiere quitar, se pone uno aleatorio. Si no lo es, no hace nada
							if (dificultad == 1) {
								dev_R[valor_evaluado] = 1 + (int)(hiprand_uniform(estado) * 4);
							}
							else {
								dev_R[valor_evaluado] = 1 + (int)(hiprand_uniform(estado) * 6);
							}
						}
					}
					else {//No se esta evaluando el ultimo valor
						if (dev_R[valor_evaluado] == numero_quitar) {//Si el valor evaluado, coincide con el que se quiere quitar... .Si no lo es, no hace nada
							//hay que coger el valor que tiene encima. Opciones: 1-El que tenga encima hay que quitarlo. 2- El que tenga encima no hay que quitarlo. Se va  ahacer un bucle que se romperá con un brake para saber qué 
							//tiene encima. Si o si va a haber gravedad.
							int valor_referencia = valor_evaluado - c; //Posicion de la matriz que se tendrá como referencia para coger su valor y aplicar gravedad
							int randoms_a_poner = 0;
							while (true) {
								if (dev_R[valor_referencia] == numero_quitar) { //Si el numero que se toma como referencia, se va a quitar a posteriori, se cogerá su valor de encima.
									valor_referencia -= c; //Para que siga avanzando por el bucle
									if (valor_referencia < 0) { //No se ha encontrado un valor de referencia, por lo que toda la fila es de numeros iguales y hay que meter todo randoms
										for (int j = valor_evaluado; j > 0; j = j - c) { //Pone a randoms todos los valores desde el evaluado hasta el final
											if (dificultad == 1) {
												dev_R[j] = 1 + (int)(hiprand_uniform(estado) * 4);
											}
											else {
												dev_R[j] = 1 + (int)(hiprand_uniform(estado) * 6);
											}
										}
										flag = 1;
										break;
									}
									randoms_a_poner += 1;
								}
								else {//Si el valor de referencia no se va a quitar, rompe el bucle, ya que ya se sabe su posicion
									randoms_a_poner += 1;
									break;
								}
							}
							//APLICAR GRAVEDAD a partir dle valor de referencia Y FLAG == 0
							if (flag == 0) { //Si no se ha llenado toda la fila de randoms, tomar el valor de VALOR_REFERENCIA para aplicar gravedad al VALOR_EVALUADO
								
								int evaluar_y_referencia = valor_evaluado; //Se usara como auxiliar para iterar desde el valor evaluado hasta el final
								for (int j = valor_referencia; j > 0; j = j - c) { //Va a aplicar gravedad a todos los valores desde el de referencia, excepto al primero, que ese siempre es aleatorio
									dev_R[evaluar_y_referencia] = dev_R[j];
									evaluar_y_referencia -= c;//Para que siga avanzando por los a apligar gravedad
								}
								int valor_columna_inicial = c - k; //Coge el valor inicial de la columna en base al numero de columnas y la iteracion en la que se encuentre k
								for (int g = 0; g < randoms_a_poner; g++) {//TIENE EN CUENTA CUANTOS VALORES HAY QUE METER ALEATORIOS, por cada numero saltado por ser igual, se pondra un numero aleaorio
									//Estos valores aleatorios iran en los primeros valores de la columna
									if (dificultad == 1) {
										dev_R[valor_columna_inicial] = 1 + (int)(hiprand_uniform(estado) * 4);
									}
									else {
										dev_R[valor_columna_inicial] = 1 + (int)(hiprand_uniform(estado) * 6);
									}
									valor_columna_inicial += c;
								}
							}
						}
					}
					valor_evaluado -= c;
				}
			}
		}
	}
}



int main(int argc, char** argv) {
	//------------------------------------- declaraciones de variables-------------------------------
	// 
	// 
	
	printf("Obtener las caracteristicas basicas de CUDA de tu tarjeta grafica.\n\n");
	int numDevices;
	hipGetDeviceCount(&numDevices);
	int minimo_bloques;
	int minimo_hilos;

	for (int i = 0; i < numDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);

		printf("Nombre del Device: %s (Numero: %d)\n", prop.name, i);
		printf("Numero maximo de hilos por bloque: %d\n", prop.maxThreadsPerBlock);
		printf("Numero maximo de hilos en un SM: %d\n", prop.maxThreadsPerMultiProcessor);
		printf("Dimensiones maximas para organizar los hilos en bloques (x,y,z): (%d,%d,%d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Dimensiones maximas para organizar los bloques en el grid (x,y,z): (%d,%d,%d)\n\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

		minimo_bloques = prop.maxGridSize[0];

		if (minimo_bloques > prop.maxGridSize[1]) {
			minimo_bloques = prop.maxGridSize[1];
		}
		
		minimo_hilos = prop.maxThreadsDim[0];
		if (minimo_hilos > prop.maxThreadsDim[1]) {
			minimo_hilos = prop.maxThreadsDim[1];
		}

	}

	//Seed para la generacion de numero aleatorios.
	srand(time(NULL));

	hiprandState* d_estado;
	hipMalloc(&d_estado, sizeof(hiprandState));

	//Para el while, si es distnto a -1 se termina el juego
	int vidas = 5;

	//Se instancia las variables que nos indica si se ha realizado algun kernel
	int bomba, par, TNT, rompeCabezas, n, explosionB, automatico, f_comprobar, c_comprobar, explosionTNT, explosionCabezas;


	//llamadas a funciones
	cout << "Introduce el numero de filas de la matriz: ";
	int filas = pedirFilas();
	cout << "Introduce el numero de columnas de la matriz: ";
	int columnas = pedirColumnas();
	int dificultad = pedirDificultad();

	int minimo_filas_columnas = columnas;
	if (filas < minimo_filas_columnas) { minimo_filas_columnas = filas; }

	int minimo_global = 0;
	if (minimo_filas_columnas <= minimo_bloques) { minimo_global = minimo_filas_columnas; }
	else { minimo_global = minimo_bloques; }

	int blockSize = 0;
	double raiz = sqrt(minimo_global);
	double redondo = round(raiz);

	blockSize = (rand() % (int)redondo) + 1;

	int tamano = ceil((filas * columnas) / (blockSize * blockSize));
	int tamano_raiz = round(sqrt(tamano))+1;

	printf("Bloques Elegidos: %d, %d\n", blockSize, blockSize);
	printf("Hilos Elegidos: %d, %d\n", tamano_raiz, tamano_raiz);


	printf("dificultad %d", dificultad);

	if (dificultad == 1) {
		n = 5;
	}
	else {
		n = 7;
	}

	printf("\n[Informacion] Desea jugar de forma manual(0) o automatico(1) \n");
	scanf("%d", &automatico);

	//Instanciacion de la matriz host y resultante
	int size = filas * columnas;
	//int* h_matriz = (int*)malloc(size* sizeof(int));
	int* h_R = (int*)malloc(size * sizeof(int));

	//---------------------------------------------------------- GENERAR MATRIZ -------------------------------------------------------------
	//UNICO METODO GENERAR MATRIZ EN h_R - OBLIGATORIO
	generarMatriz(h_R, filas, columnas, dificultad, d_estado); // XXXXXXXXXX - USAAAAAAAAAAAAAAAAR
	//MATRIZ EJEMPLO
	/*
	int test_matriz[10][9]= {{1, 1, 1, 1, 1, 5, 5, 5, 5},
							 {2, 2, 2, 2, 2, 2, 6, 6, 6},
							 {3, 3, 1, 6, 1, 4, 3, 3, 3},
							 {4, 2, 4, 5, 1, 4, 4, 4, 4},
							 {1, 5, 4, 5, 1, 4, 5, 5, 5},
							 {1, 6, 1, 5, 1, 4, 6, 6, 6},
							 {1, 1, 1, 5, 1, 4, 1, 1, 1},
							 {1, 2, 1, 5, 1, 4, 2, 2, 3},
							 {1, 6, 6, 5, 6, 6, 6, 3, 4},
							 {1, 4, 4, 5, 4, 4, 4, 3, 4}};
	for (int i = 0; i < 10; i++) { // XXXXXXXXXX - BORRAR CUANDO NO SE QUIERA USAR UNA MATRIZ PREDEFINIDA DE EJEMPLO
		for (int j = 0; j < 9; j++) { // XXXXXXXXXX - BORRAR CUANDO NO SE QUIERA USAR UNA MATRIZ PREDEFINIDA DE EJEMPLO
			h_R[i * 9 + j] = test_matriz[i][j]; // XXXXXXXXXX - BORRAR CUANDO NO SE QUIERA USAR UNA MATRIZ PREDEFINIDA DE EJEMPLO
		} // XXXXXXXXXX - BORRAR CUANDO NO SE QUIERA USAR UNA MATRIZ PREDEFINIDA DE EJEMPLO
	} // XXXXXXXXXX - BORRAR CUANDO NO SE QUIERA USAR UNA MATRIZ PREDEFINIDA DE EJEMPLO*/
	//---------------------------------------------------------------------------------------------------------------------------------------
	
	//int h_matriz[] = { 1,1,1,1,0,2,4,2,3,3,2,1,2,3,0,0,1,1,2,3,1,2,1,3,3 };

	//instanciamos las variables del Device
	int* dev_R, * d_bomba, * d_par, * d_TNT, * d_rompeCabezas, * d_explosionB, * d_explosionTNT, *d_explosionCabezas;
	//int* dev_M; NO ES NECESARIO NO SE USA

	//Numero aleatorios ene la GPU
	

	//-------------------------------------cuestiones de memoria-------------------------------

	//hipMalloc((void**)&dev_M, size * sizeof(int)); NO ES NECESARIO NO SE USA
	// Reservamos memoria en la GPU
	hipMalloc((void**)&dev_R, size * sizeof(int));


	hipMalloc((void**)&d_par, sizeof(int));
	hipMalloc((void**)&d_bomba, sizeof(int));
	hipMalloc((void**)&d_TNT, sizeof(int));
	hipMalloc((void**)&d_rompeCabezas, sizeof(int));
	hipMalloc((void**)&d_explosionB, sizeof(int));
	hipMalloc((void**)&d_explosionTNT, sizeof(int));
	hipMalloc((void**)&d_explosionCabezas, sizeof(int));

	// Copiamos los datos desde la memoria host a la memoria device
	hipMemcpy(dev_R, h_R, size * sizeof(int), hipMemcpyHostToDevice);


	//definimos el numero de bloques y hilos por bloque
	//dim3 numBloques(1);
	//dim3 ThreadsBloque(filas, filas);

	dim3 numBloques(blockSize, blockSize);
	dim3 ThreadsBloque(tamano_raiz, tamano_raiz);

	//Se copia la matriz inicial en la GPU - NO ES NECESARIO NO SE USA
	//copiarMatriz <<<numBloques, ThreadsBloque>>> (dev_M, dev_R, columnas); NO ES NECESARIO NO SE USA


	while (vidas > 0) {
		//se resetean los comprobadores
		bomba = 0, par = 0, TNT = 0, rompeCabezas = 0, explosionB = 0, explosionTNT = 0, explosionCabezas = 0;

		hipMemcpy(d_par, &par, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_bomba, &bomba, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_explosionB, &explosionB, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_TNT, &TNT, sizeof(int), hipMemcpyHostToDevice);	
		hipMemcpy(d_explosionTNT, &explosionTNT, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_rompeCabezas, &rompeCabezas, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_explosionCabezas, &explosionCabezas, sizeof(int), hipMemcpyHostToDevice);

		hipMemcpy(dev_R, h_R, size * sizeof(int), hipMemcpyHostToDevice);

		//se imprime la matriz mostrando cada cambio
		imprimirMatriz2(h_R, columnas, filas);

		if (automatico == 0) {
			printf("\nIntroduce la fila que quiera comprobar: ");
			f_comprobar = pedirFilas();
			printf("\nIntroduce la columna que quiera comprobar: ");
			c_comprobar = pedirColumnas();
		}
		else {
			f_comprobar = rand() % filas;
			c_comprobar = rand() % columnas;

			printf("\nSe juega en la fila y columna: %d %d", f_comprobar, c_comprobar);
		}




		//------------------------------------- kernels -------------------------------

		//KERNEL-> EXPLOTAR ROMPECABEZAS
		explotarRompecabezas<<< numBloques, ThreadsBloque >>> (dev_R, columnas, filas, size, f_comprobar, c_comprobar, d_explosionCabezas, dificultad, d_estado);
		hipMemcpy(&explosionCabezas, d_explosionCabezas, sizeof(int), hipMemcpyDeviceToHost);
		if (explosionCabezas == 0) {
			comprobarRompecabezas << <numBloques, ThreadsBloque >> > (dev_R, columnas, filas, size, f_comprobar, c_comprobar, d_rompeCabezas, dificultad, d_estado);
			hipMemcpy(&rompeCabezas, d_rompeCabezas, sizeof(int), hipMemcpyDeviceToHost);
		}
		if (rompeCabezas == 0 && explosionCabezas == 0) {
			explotarTNT << <numBloques, ThreadsBloque >> > (dev_R, columnas, filas, size, f_comprobar, c_comprobar, n, d_explosionTNT, dificultad, d_estado);
			hipMemcpy(&explosionTNT, d_explosionTNT, sizeof(int), hipMemcpyDeviceToHost);
		}
		if (rompeCabezas == 0 && explosionCabezas == 0 && explosionTNT == 0) {
			ponerTNT << <numBloques, ThreadsBloque >> > (dev_R, columnas, filas, size, f_comprobar, c_comprobar, d_TNT, n, dificultad, d_estado);
			hipMemcpy(&TNT, d_TNT, sizeof(int), hipMemcpyDeviceToHost);
		}
		if (rompeCabezas == 0 && explosionCabezas == 0 && explosionTNT == 0 && TNT == 0) {
			explotarBomba << <numBloques, ThreadsBloque >> > (dev_R, columnas, filas, size, f_comprobar, c_comprobar, n, d_explosionB, dificultad, d_estado);
			hipMemcpy(&explosionB, d_explosionB, sizeof(int), hipMemcpyDeviceToHost);
		}
		if (rompeCabezas == 0 && explosionCabezas == 0 && explosionTNT == 0 && TNT == 0 && explosionB == 0) {
			ponerBomba << <numBloques, ThreadsBloque >> > (dev_R, columnas, filas, size, f_comprobar, c_comprobar, d_bomba, n, dificultad, d_estado);
			hipMemcpy(&bomba, d_bomba, sizeof(int), hipMemcpyDeviceToHost);
		}
		if (rompeCabezas == 0 && explosionCabezas == 0 && explosionTNT == 0 && TNT == 0 && explosionB == 0 && bomba == 0) {	
			comprobarPares << <numBloques, ThreadsBloque >> > (dev_R, columnas, filas, size, f_comprobar, c_comprobar, d_par, dificultad, d_estado);
			hipMemcpy(&par, d_par, sizeof(int), hipMemcpyDeviceToHost);
		}
	
		hipMemcpy(h_R, dev_R, size * sizeof(int), hipMemcpyDeviceToHost);

		if (par == 0 && bomba == 0 && TNT == 0 && rompeCabezas == 0 && explosionB == 0 && explosionTNT == 0 && explosionCabezas == 0) {
			printf("\n[Informacion] Se ha perdido una vida:\n");
			printf("\nNo se ha podido realizar ninguna accion en las coordenadas: %d %d\n", f_comprobar, c_comprobar);
			vidas = vidas - 1;
			printf("\nVidas restantes: %d\n", vidas);
		}


	}

	//---------------------------- Fin de la programacion paralela---------------------------------------
	// copiando el resultado a la memoria Host
	hipMemcpy(h_R, dev_R, size * sizeof(int), hipMemcpyDeviceToHost);



	printf("\nFin del Juego!!!!!\n");
	printf("**********************");
	printf("\n La matriz Resultante es:\n");
	imprimirMatriz2(h_R, columnas, filas);
	//hipFree(dev_M);
	hipFree(dev_R);

	hipFree(d_bomba);
	hipFree(d_par);
	hipFree(d_TNT);
	hipFree(d_rompeCabezas);


	return (EXIT_SUCCESS);
}