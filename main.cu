#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <fstream>
using namespace std;

//conjunto de 4 numeros localizados en memoria

const int Col = 6;
const int Fil = 6;
const int SIZE = Fil*Col;



__global__ void imprimirMatriz(int *dev_A, int *dev_R) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * Col + col;
	if (pos == 0){
		printf("\n Matriz:\n");
	}
	if ((pos%Col) == 0){
		printf("\n%d",dev_R[pos]);
		printf("  ");
	}else{
		printf("%d",dev_R[pos]);
		printf("  ");
	}
	
}

__global__ void copiarMatriz(int *dev_A, int *dev_R) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * Col + col;
	dev_R[pos] = dev_A[pos];

}
	

__global__ void moverPosiciones(int *dev_A, int *dev_R, int x, int y, int dirr) {
	
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * Col + col;
	int posCambiar = x * Col + y;
	
	if (posCambiar < SIZE && posCambiar >= 0){ //se comprueba que la coor esta dentro de los limites
		if (pos == posCambiar){ //entra cuando esta en la posicion que se quiere cambiar
			if (dirr == 0 && posCambiar-col > 0 ){//indica la dirrecion 0 = arriba
				int aux =  dev_A[pos-Col];
				dev_R[pos] = aux;
				dev_R[pos-Col] = dev_A[pos];

			}
			if (dirr == 1 && (posCambiar+1)%Col !=0 ){//indica la dirrecion 1 = derecha. Si el resto +1 = 0 se trata de la ultima columna por lo que no puede moverse a la der
				int aux =  dev_A[pos+1];
				dev_R[pos] = aux;
				dev_R[pos+1] = dev_A[pos];

			}
			if (dirr == 2 && posCambiar+col < SIZE ){//indica la dirrecion 2 = abajo
				int aux =  dev_A[pos+Col];
				dev_R[pos] = aux;
				dev_R[pos+Col] = dev_A[pos];

			}
			if (dirr == 3 && (posCambiar)%Col !=0 ){//indica la dirrecion 3 = izquierda. Si el resto = 0 se trata de la primera columna por lo que no puede moverse a la izq
				int aux =  dev_A[pos-1];
				dev_R[pos] = aux;
				dev_R[pos-1] = dev_A[pos];
			}
		}
	}
}

//se busca donde haya cuatro elementos iguales, tanto en posiciones horizontales como verticales y se pone la bomba
//se tiene que guardar si es columna o fila, para despues explotar
__global__ void ponerBomba(int *dev_R) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * Col + col;
	int valor = dev_R[pos];
	int fila = pos/Col;

	//Derecha
	//compruba que los 3 elementos siguientes tienen el mismo valor y que estan en la misma fila
	if (dev_R[pos+1] == valor && dev_R[pos+2] == valor && dev_R[pos+3] == valor && pos+1/Col == fila && pos+2/Col == fila && pos+3/Col == fila){
		dev_R[pos] = -1;
		dev_R[pos+1] = rand() % 5;
		dev_R[pos+2] = rand() % 5;
		dev_R[pos+3] = rand() % 5;

	}

	//Arriba y Abajo
	//compruba que los 3 elementos de arriba tienen el mismo valor y que esta dentro de matriz
	
	if (pos+(Col*3) < SIZE){
		if (dev_R[pos+Col] == valor && dev_R[pos+(Col*2)] == valor && dev_R[pos+(Col*3)]== valor){
			dev_R[pos] = -1;
			dev_R[pos+(Col*1)] = rand() % 5;
			dev_R[pos+(Col*2)] = rand() % 5;
			dev_R[pos+(Col*3)] = rand() % 5;
		}
	}

	

}


//Chequea si hay elementos adyacentes con el mismo valor en toda la matriz, si es asi, los cambia por valores aleatorios
__global__ void comprobarPares(int *dev_R, int f, int c) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * Col + col;
	//posicion indicada por el usuario
	int posUsuario = f * Col + c;
		
	//chequea si en la posicion de de la derecha hay elementos iguales y si es la posicion indicada por el usuario
	if ( dev_R[pos] == dev_R[pos+1] && (pos+1)%Col !=0 && pos == posUsuario ){ 
		dev_R[pos] = rand() % 5;
		dev_R[pos+1] = rand() % 5;
	}

	//chequea abajo
	if ( dev_R[pos] == dev_R[pos+Col] && pos < SIZE && pos == posUsuario){
		dev_R[pos] = rand() % 5;
		dev_R[pos+Col] = rand() % 5; 
	}	
}

	

int main(int argc, char** argv) {
	//------------------------------------- declaraciones de variables-------------------------------
	//Seed para la generacion de numero aleatorios.
    srand(time(NULL));

    //Declaracion de varibles
    int filas, columnas, dificultad;
    int** h_matriz; //Matriz del host
    bool validarEnteroCheck = false;

	//declaramos la matriz
	int h_M[Fil][Col];
	
	//declaramos la lista con los valores
	int h_valores[6] = {1,2,3,4,5,6};
	
	//se instancia la matriz resultante
	int h_R[Fil][Col] = { 0 };
	

	//Para el while, si es distnto a -1 se termina el juego
	int ganador = -1; 

	//Se genera la matriz
	printf("\n Matriz Inicial:\n");
	for (int i = 0; i < Fil; i++){
		for (int j = 0; j < Col; j++) {
			h_M[i][j] = rand() % 5; //numero aleatorio entre el 0-10
			printf("%d", h_M[i][j]);
			printf("  ");
		}
		printf("\n");
	}
	printf("\n");

	//-------------------------------------cuestiones de memoria-------------------------------
	//instanciamos como vectores la matriz
	int*dev_M,*dev_valores,*dev_R;

	
	// Reservamos memoria en la GPU
	hipMalloc((void**) &dev_M, SIZE * sizeof(int));
	hipMalloc((void**) &dev_R, SIZE * sizeof(int));
	
	
	

	// Copiamos los datos desde la memoria host a la memoria device
	hipMemcpy(dev_M, h_M, SIZE* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_R, h_R, SIZE* sizeof(int), hipMemcpyHostToDevice);

	
	
	//definimos el numero de bloques y hilos por bloque
	dim3 numBloques (1); 
	dim3 ThreadsBloque(Fil, Col);
	
	//------------------------------------- kernels -------------------------------
	//pedir coordenadas de la ficha que se quiera mover y direccion 
	
	//Se copia la matriz inicial en la final
	copiarMatriz <<<numBloques, ThreadsBloque >>> (dev_M,dev_R);
	
	//moverPosiciones <<<numBloques, ThreadsBloque >>> (dev_M,dev_R,0,0,2);
	
	comprobarPares <<<numBloques, ThreadsBloque >>> (dev_R, 2,4);
	imprimirMatriz <<<numBloques, ThreadsBloque >>> (dev_M,dev_R);
	
	ponerBomba <<<numBloques, ThreadsBloque >>> (dev_R);
	
	
	


	
	
	// copiando el resultado a la memoria Host
	hipMemcpy(h_R, dev_R, SIZE* sizeof(int), hipMemcpyDeviceToHost);
	

	hipFree(dev_M);
	hipFree(dev_R);
	
	printf("\n");
	printf("El resultado de mover posiciones de las matrices:\n");
	for (int i = 0; i < Fil; i++) {
		for (int j = 0; j < Col; j++) {
			printf("%d",h_R[i][j]);
			printf("  ");
			

		}
		printf("\n");
			
	} 

	return (EXIT_SUCCESS);
}