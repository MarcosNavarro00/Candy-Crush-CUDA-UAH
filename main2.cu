#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <fstream>
using namespace std;



//altan
//Bomba
//tnt
//gravedad
//rompecabezas
//Automatico




//---------M�todo que comprueba si lo introducido por consola es un valor entero.---------
bool validarEntero(int& numero) {
    //Si no es un numero entero lo introducido, el c�digo gestiona el error.
    if (cin.fail()) {
        cin.clear();
        cin.ignore();
        cout << "[Error]: El valor introducido no es correcto." << endl;
        cout << endl;
        return false;
    }
    else {
        cout << endl;
        return true;
    }
}

//---------M�todo que pide al usuario las filas para comenzar.---------
int pedirFilas(){
	int filas;
    bool validarEnteroCheck = false;

	while (!validarEnteroCheck) {
        cin >> filas;
        validarEnteroCheck = validarEntero(filas);
    }
    validarEnteroCheck = false;
	return filas;
}

//---------M�todo que pide al usuario las columnas para comenzar.---------
int pedirColumnas(){
	int columnas;
    bool validarEnteroCheck = false;
	//--------Pide las columnas de la matriz. Si no introduce un numero entero, error.------------
    while (!validarEnteroCheck) {
        cin >> columnas;
        validarEnteroCheck = validarEntero(columnas);
    }
    validarEnteroCheck = false;
	return columnas;
}

//---------M�todo que pide al usuario la dificultad del juego.---------
int pedirDificultad(){
	//Pide el nivel de dificultad. Si no introduce un numero entero o (1 o 2),Error.
    // FACIL = 1. DIFICIL = 2
	 int dificultad;
    bool validarEnteroCheck = false;
    while (!validarEnteroCheck) {
        cout << "Selecciona nivel de dificultad (1 = FACIL | 2 = DIFICIL)" << endl;
        cout << "\tFACIL -> Candry Crosh genera numeros del 1 al 4." << endl;
        cout << "\tDIFICIL -> Candry Crosh genera numeros del 1 al 6." << endl;
        cout << "Introduce el numero: ";
        cin >> dificultad;
        validarEnteroCheck = validarEntero(dificultad);
        if (dificultad != 1 && dificultad != 2) { validarEnteroCheck = false; }
    }
    validarEnteroCheck = false;
	return dificultad;

}

//---------M�todo que genera la matriz inicial y la devuelve.---------
int* generarMatriz (int* h_matriz,int filas, int columnas, int dificultad){
	//Se genera la matriz
	printf("\n Da comienzo el juego\n");
	printf("**********************");
	printf("\n Se genera aleatoriamente la siguiente matriz:\n");
	for (int i = 0; i < filas; i++){
		for (int j = 0; j < columnas; j++) {
			if (dificultad == 1) {
				h_matriz[i*columnas+j]= rand() % 5; //numero aleatorio entre el 0-10
				printf("%d", h_matriz[i*columnas+j]);
				printf("  ");
			} else if (dificultad == 2) {
                h_matriz[i*columnas+j] = rand() % 7;
				printf("%d", h_matriz[i*columnas+j]);
				printf("  ");
            }       

		}
		printf("\n");
	}
	printf("**********************\n");

	return h_matriz;

}

//---------M�todo que genera la matriz mientras el juego esta en marcha.---------
__global__ void imprimirMatriz(int *dev_R, int c) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * c + col;
	if (pos == 0){
		printf("\n Matriz:\n");
	}
	if ((pos%c) == 0){
		printf("\n%d",dev_R[pos]);
		printf("  ");
	}else{
		printf("%d",dev_R[pos]);
		printf("  ");
	}
	
}

//---------M�todo que copia la matriz de dev_A a dev_R.---------
__global__ void copiarMatriz(int *dev_A, int *dev_R, int c) {
	//declaramos las col y filas
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = i * c + j;
	dev_R[pos] = dev_A[pos];

}
	
/*
__global__ void moverPosiciones(int *dev_A, int *dev_R, int x, int y, int dirr) {
	
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * Col + col;
	int posCambiar = x * Col + y;
	
	if (posCambiar < SIZE && posCambiar >= 0){ //se comprueba que la coor esta dentro de los limites
		if (pos == posCambiar){ //entra cuando esta en la posicion que se quiere cambiar
			if (dirr == 0 && posCambiar-col > 0 ){//indica la dirrecion 0 = arriba
				int aux =  dev_A[pos-Col];
				dev_R[pos] = aux;
				dev_R[pos-Col] = dev_A[pos];

			}
			if (dirr == 1 && (posCambiar+1)%Col !=0 ){//indica la dirrecion 1 = derecha. Si el resto +1 = 0 se trata de la ultima columna por lo que no puede moverse a la der
				int aux =  dev_A[pos+1];
				dev_R[pos] = aux;
				dev_R[pos+1] = dev_A[pos];

			}
			if (dirr == 2 && posCambiar+col < SIZE ){//indica la dirrecion 2 = abajo
				int aux =  dev_A[pos+Col];
				dev_R[pos] = aux;
				dev_R[pos+Col] = dev_A[pos];

			}
			if (dirr == 3 && (posCambiar)%Col !=0 ){//indica la dirrecion 3 = izquierda. Si el resto = 0 se trata de la primera columna por lo que no puede moverse a la izq
				int aux =  dev_A[pos-1];
				dev_R[pos] = aux;
				dev_R[pos-1] = dev_A[pos];
			}
		}
	}
}

*/
//se busca donde haya cuatro elementos iguales, tanto en posiciones horizontales como verticales y se pone la bomba
//se tiene que guardar si es columna o fila, para despues explotar

//---------M�todo que pone la bomba donde haya 4 elementos iguales, indicados por el usuario.---------
__global__ void ponerBomba(int *dev_R, int c, int size, int fila, int columna) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * c + col;
	int valor = dev_R[pos];
	int fila_ini = pos/c;

	//Derecha
	//compruba que los 3 elementos siguientes tienen el mismo valor y que estan en la misma fila
	if (dev_R[pos+1] == valor && dev_R[pos+2] == valor && dev_R[pos+3] == valor && pos+1/c == fila_ini && pos+2/c == fila && pos+3/c == fila_ini){
		dev_R[pos] = -1;
		dev_R[pos+1] = rand() % 5;
		dev_R[pos+2] = rand() % 5;
		dev_R[pos+3] = rand() % 5;

	}

	//Arriba y Abajo
	//compruba que los 3 elementos de arriba tienen el mismo valor y que esta dentro de matriz
	
	if (pos+(c*3) < size){
		if (dev_R[pos+c] == valor && dev_R[pos+(c*2)] == valor && dev_R[pos+(c*3)]== valor){
			dev_R[pos] = -1;
			dev_R[pos+(c*1)] = rand() % 5;
			dev_R[pos+(c*2)] = rand() % 5;
			dev_R[pos+(c*3)] = rand() % 5;
		}
	}

	

}


//Chequea si hay elementos adyacentes con el mismo valor en toda la matriz, si es asi, los cambia por valores aleatorios
__global__ void comprobarPares(int *dev_R, int fila, int columna, int c, int size) {
	//declaramos las col y filas
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int fil = threadIdx.y + blockDim.y * blockIdx.y;
	
	//posicion de la matriz final 
	int pos = fil * c + col;
	//posicion indicada por el usuario
	int posUsuario = fila * c + columna;
		
	//chequea si en la posicion de de la derecha hay elementos iguales y si es la posicion indicada por el usuario
	if ( dev_R[pos] == dev_R[pos+1] && (pos+1)%c !=0 && pos == posUsuario ){ 
		dev_R[pos] = rand() % 5;
		dev_R[pos+1] = rand() % 5;
	}

	//chequea abajo
	if ( dev_R[pos] == dev_R[pos+c] && pos < size && pos == posUsuario){
		dev_R[pos] = rand() % 5;
		dev_R[pos+c] = rand() % 5; 
	}	
}

	

int main(int argc, char** argv) {
	//------------------------------------- declaraciones de variables-------------------------------
	//Seed para la generacion de numero aleatorios.
    //srand(time(NULL));
	
	//Para el while, si es distnto a -1 se termina el juego
	int ganador = -1; 
	
	//llamadas a funciones
	cout << "Introduce el numero de filas de la matriz: ";
	int filas=pedirFilas();
	cout << "Introduce el numero de columnas de la matriz: ";
	int columnas = pedirColumnas();
	int dificultad = pedirDificultad();

	//Instanciacion de la matriz host y resultante
	int size = filas * columnas ;
	int* h_matriz = (int*)malloc(size* sizeof(int));
	int* h_R = (int*)malloc(size* sizeof(int));

	h_matriz = generarMatriz (h_matriz,filas,columnas, dificultad);

	//-------------------------------------cuestiones de memoria-------------------------------
	//instanciamos como vectores la matriz
	int*dev_M,*dev_R;

	// Reservamos memoria en la GPU
	hipMalloc((void**) &dev_M, size * sizeof(int));
	hipMalloc((void**) &dev_R, size * sizeof(int));
	
	// Copiamos los datos desde la memoria host a la memoria device
	hipMemcpy(dev_M, h_matriz, size* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_R, h_R, size* sizeof(int), hipMemcpyHostToDevice);

	
	//definimos el numero de bloques y hilos por bloque
	dim3 numBloques (1); 
	dim3 ThreadsBloque(filas, columnas);


	while(ganador==-1){
		printf("\n Introduce la fila y columna que quiera comprobar:\n");
		int f_comprobar=pedirFilas();
		int c_comprobar = pedirColumnas();
		//------------------------------------- kernels -------------------------------
		//pedir coordenadas de la ficha que se quiera mover y direccion 
		
		//Se copia la matriz inicial en la final
		copiarMatriz <<<numBloques, ThreadsBloque >>> (dev_M,dev_R, columnas);
		ponerBomba <<<numBloques, ThreadsBloque >>> (dev_R,columnas,size,f_comprobar,c_comprobar);
		
		//moverPosiciones <<<numBloques, ThreadsBloque >>> (dev_M,dev_R,0,0,2);
		
		comprobarPares <<<numBloques, ThreadsBloque >>> (dev_R, f_comprobar,c_comprobar,columnas,size);
		imprimirMatriz <<<numBloques, ThreadsBloque >>> (dev_R,columnas);
		
	
	}
	
	//---------------------------- Fin de la programacion paralela---------------------------------------
	// copiando el resultado a la memoria Host
	hipMemcpy(h_R, dev_R, size* sizeof(int), hipMemcpyDeviceToHost);
	
	
	printf("\nFin del Juego!!!!!\n");
	printf("**********************");
	printf("\n La matriz Resultante es:\n");
	for (int i = 0; i < filas; i++) {
		for (int j = 0; j < columnas; j++) {
			printf("%d",h_R[i*columnas+j]);
			printf("  ");
			

		}
		printf("\n");
			
	} 
	hipFree(dev_M);
	hipFree(dev_R);

	return (EXIT_SUCCESS);
}